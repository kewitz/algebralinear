/*
 * The MIT License (MIT)
 *
 * Copyright (c) 2014 Leonardo Kewitz
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>



#define DEBUG true

__global__ void JacobiIter(int n, int k, double* A, double* x, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x, j;
    if (i >= n) return;

    int off0 = ((k % 2) * n);
    int off1 = n - off0;

    double w = b[i];
    for (j = 0; j < i; j++) {
        w -= A[i*n + j] * x[j + off0];
    }
    for (j = i+1; j < n; j++) {
        w -= A[i*n + j] * x[j + off0];
    }
    w /= A[i*n + i];
    x[i + off1] = w;
}

extern "C" void CUJacobi(int n, int ks, double* A, double* x, double* b) {
    int k;

    double *dA, *dx, *db;
    if (DEBUG) printf("[+] CUDA Malloc...\n");
    hipMalloc(&dA, sizeof(double)*n*n);
    hipMalloc(&dx, sizeof(double)*n*2);
    hipMalloc(&db, sizeof(double)*n);

    if (DEBUG) printf("[+] Copying to device memory...\n");
    hipMemcpy(dA, A, sizeof(double)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(dx, x, sizeof(double)*n, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(double)*n, hipMemcpyHostToDevice);

    const dim3 threads(64, 1);
    const dim3 blocks(1 + n/64, 1);
    if (DEBUG) printf("[+] Running Kernel...\n");
    for (k = 0; k < ks; k++) {
        JacobiIter<<<blocks, threads>>>(n, k, dA, dx, db);
        hipDeviceSynchronize();
    }
    if (DEBUG) printf("[+] Copying Result and freeing memory...\n");
    hipMemcpy(x, dx, sizeof(double)*n, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dx);
    hipFree(db);
    if (DEBUG) printf("[+] Done.\n");
    return;
}
